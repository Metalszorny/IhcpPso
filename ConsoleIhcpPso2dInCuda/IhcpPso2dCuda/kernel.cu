#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include <algorithm>
#include <time.h>
#include <iomanip>
#include <iterator>
#include <thread>

#pragma region Fields

double* _initialTemperature;
int* _horizontalSplitting;
int* _verticalSplitting;
int* _monitoredIndex;
double* _timeDifference;
int* _dimensionNumber;
double* _htcValues;
double* _specificHeats;
int* _specificHeatsSizeRows;
double* _heatConductivities;
int* _heatConductivitiesSizeRows;
int* _rangeMin;
int* _rangeMax;
double* _simulationTime;
double* _height;
double* _radius;
double* _density;
double* _dX;
double* _dY;
double* _referenceCooldownCurve;
int* _referenceCooldownCurveSizeRows;
double* _tK;
double* _particleInitialVelocity;
int* _particleInitialVelocitySize;
double* _particleInitialPosition;
int* _particleInitialPositionSize;
int* _particleInformerNumber;
double* _particleConstant1;
double* _particleConstant2;
double* _particleEpsilon;
double* _particlePosition;
double* _particleVelocity;
double* _particleBestPosition;
double* _particleFitness;
double* _particleBestFitness;
double* _globalBestFitness;
double* _globalBestPosition;
int* _globalBestSize;
int* _particleOptimalisationType;
int* _particleSwarmSize;
int* _maxEpochs;
int* _maxStaticEpochs;
double* _weight;
int* _particleInformers;
std::string _exitReason;
int* _epoch;
hipError_t _cudaStatus;
std::string _cudaError;

#pragma endregion

#pragma region Methods

void ReadDataFromFile(std::string dataFilePath);
void CalculateReferenceCooldownCurve();
double* CalculateCooldownCurve1D(bool isParticle, int particleIndex, double* currentTemperature, double* previousTemperature, double* g);
double GetHeatConductivity(double temperature);
double GetSpecificHeat(double temperature);
double GetAlpha(double heatConductivity, double specificHeat);
double GetHeatTransferCoefficient(double temperature, bool isParticle, int particleIndex);
void WriteReferenceCooldownLogToFile();
void WritePsoGlobalBestLogToFile();
void WriteExitResultAndTimeLogToFile(long elaspedMilliseconds);
void WritePreIterationResult();
void WriteIterationResult(int iteration);
void WriteCudaErrorToFile();
void Optimize();
void UpdateRing();
int* GetIntegerRange(int startIndex, int count);
int* Shuffle(int* index, int indexSize);
double GetSwarmAverageBestPosition();
std::vector<std::string> explode(std::string originalString, char delimeter);
template<typename Out> void split(std::string s, char delim, Out result);
void InitializeParticle();
void OptimizePosition(double* beta, double* mBest);

__global__ void InitializeParticleKernel(int* dimensionNumber, double* particlePosition,
	double* particleVelocity, double* particleInitialPosition, int* particleInitialPositionSize,
	double* particleInitialVelocity, int* particleInitialVelocitySize, int* rangeMin, int* rangeMax,
	double* randomNumbers1, double* randomNumbers2)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Set initial position.
	for (int i = 0; i < dimensionNumber[0]; i += 1)
	{
		// The given values are null.
		if (particleInitialPosition == NULL)
		{
			particlePosition[(index * dimensionNumber[0]) + i] = ((rangeMax[0] - rangeMin[0]) * randomNumbers1[(index * dimensionNumber[0]) + i] + rangeMin[0]);
		}
		// There is only one given value.
		else if (particleInitialPositionSize[0] == 1)
		{
			particlePosition[(index * dimensionNumber[0]) + i] = particleInitialPosition[0];
		}
		// There are as many given value as the position dimensions.
		else if (particleInitialPositionSize[0] == dimensionNumber[0])
		{
			particlePosition[(index * dimensionNumber[0]) + i] = particleInitialPosition[i];
		}
		// The current position can be set from the given values.
		else if (i < particleInitialPositionSize[0])
		{
			particlePosition[(index * dimensionNumber[0]) + i] = particleInitialPosition[i];
		}
		// The current position can't be set from the given values.
		else
		{
			particlePosition[(index * dimensionNumber[0]) + i] = ((rangeMax[0] - rangeMin[0]) * randomNumbers1[(index * dimensionNumber[0]) + i] + rangeMin[0]);
		}
	}

	// Set initial velocity.
	for (int i = 0; i < dimensionNumber[0]; i += 1)
	{
		// The given values are null.
		if (particleInitialVelocity == NULL)
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = ((rangeMax[0] - rangeMin[0]) * randomNumbers2[(index * dimensionNumber[0]) + i] + rangeMin[0]);
		}
		// There is only one given value.
		else if (particleInitialVelocitySize[0] == 1)
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = particleInitialVelocity[0];
		}
		// There are as many given value as the velocity dimensions.
		else if (particleInitialVelocitySize[0] == dimensionNumber[0])
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = particleInitialVelocity[i];
		}
		// The current velocity can be set from the given values.
		else if (i < particleInitialVelocitySize[0])
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = particleInitialVelocity[i];
		}
		// The current velocity can't be set from the given values.
		else
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = ((rangeMax[0] - rangeMin[0]) * randomNumbers2[(index * dimensionNumber[0]) + i] + rangeMin[0]);
		}
	}
}

__global__
void OptimizePositionKernel(int* particleOptimalisationType, double* mbest, double* beta,
	int* dimensionNumber, int* particleInformerNumber, int* particleInformers, double* particleBestFitness,
	double* particleBestPosition, double* weight, double* particleVelocity, double* particlePosition,
	int* rangeMin, int* rangeMax, int* globalBestSize, double* globalBestPosition, double* particleConstant1,
	double* particleConstant2, double* randomNumbers1, double* randomNumbers2, double* randomNumbers3,
	double* bestLocalPosition)
{
	// Get the iteration number.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Update the particle's position based on the type of the PSO.
	switch (particleOptimalisationType[0])
	{
		// Cleck.
	case 1:
		int bestIndex = index;

		// Get the minimum fitness from the particle or it's informers.
		for (int i = 0; i < particleInformerNumber[0]; i += 1)
		{
			if (particleBestFitness[particleInformers[(index * particleInformerNumber[0]) + i]] < particleBestFitness[bestIndex])
			{
				bestIndex = particleInformers[(index * particleInformerNumber[0]) + i];
			}
		}

		// Set the potinion of the found minimum fitness particle.
		for (int i = 0; i < dimensionNumber[0]; i += 1)
		{
			bestLocalPosition[i] = particleBestPosition[(bestIndex * dimensionNumber[0]) + i];
		}

		// Update the particle's velocity.
		for (int i = 0; i < dimensionNumber[0]; i += 1)
		{
			particleVelocity[(index * dimensionNumber[0]) + i] = (weight[0] * particleVelocity[(index * dimensionNumber[0]) + i]) +
				(particleConstant1[0] * randomNumbers1[(index * dimensionNumber[0]) + i] * (particleBestPosition[(index * dimensionNumber[0]) + i] -
					particlePosition[(index * dimensionNumber[0]) + i])) + (particleConstant2[0] * randomNumbers2[(index * dimensionNumber[0]) + i] *
					(bestLocalPosition[i] - particlePosition[(index * dimensionNumber[0]) + i]));
		}

		// Update the particle's position.
		for (int i = 0; i < dimensionNumber[0]; i += 1)
		{
			particlePosition[(index * dimensionNumber[0]) + i] = (particlePosition[(index * dimensionNumber[0]) + i] + particleVelocity[(index * dimensionNumber[0]) + i]);
		}
		break;
		// Quantum.
	case 2:
		// Update the particle's position.
		for (int i = 0; i < dimensionNumber[0]; i += 1)
		{
			double fi = randomNumbers3[(index * dimensionNumber[0]) + i];
			double p = fi * particleBestPosition[(index * dimensionNumber[0]) + i] + (1 - fi) * globalBestPosition[(globalBestSize[0] - 1 - dimensionNumber[0]) + i];

			if (fi > 0.5)
			{
				particlePosition[(index * dimensionNumber[0]) + i] = p - beta[0] * fabs(mbest[0] - particlePosition[(index * dimensionNumber[0]) + i]) * (-log10(fi));
			}
			else
			{
				particlePosition[(index * dimensionNumber[0]) + i] = p + beta[0] * fabs(mbest[0] - particlePosition[(index * dimensionNumber[0]) + i]) * (-log10(fi));
			}

			if (particlePosition[(index * dimensionNumber[0]) + i] < rangeMin[0])
			{
				particlePosition[(index * dimensionNumber[0]) + i] = 2 * rangeMin[0] - particlePosition[(index * dimensionNumber[0]) + i];
			}

			if (particlePosition[(index * dimensionNumber[0]) + i] > rangeMax[0])
			{
				particlePosition[(index * dimensionNumber[0]) + i] = 2 * rangeMax[0] - particlePosition[(index * dimensionNumber[0]) + i];
			}
		}
		break;
	}
}

__global__
void ObjectiveFunctionKernel(int* dimensionNumber, double* particleBestFitness,
	double* particleBestPosition, double* particlePosition,
	int* rangeMin, int* rangeMax, double* particleFitness, double* initialTemperature, int* horizontalSplitting,
	int* referenceCooldownCurveSizeRows, double* referenceCooldownCurve, int* monitoredIndex,
	double* timeDifference, double* dX, double* tK, int* heatConductivitiesSizeRows, double* heatConductivities,
	int* specificHeatsSizeRows, double* specificHeats, double* density, double* htcValues,
	double* currentTemperature, double* previousTemperature, double* g)
{
	// Get the iteration number.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	bool isInRange = true;

	// Check if the position of the particle is the specified range.
	for (int i = 0; i < dimensionNumber[0]; i += 1)
	{
		if (particlePosition[(index * dimensionNumber[0]) + i] < rangeMin[0] ||
			particlePosition[(index * dimensionNumber[0]) + i] > rangeMax[0])
		{
			isInRange = false;
		}
	}

	// The position of the particle is in the specified range.
	if (isInRange)
	{
		particleFitness[index] = 0;

		// Simulate cooldown and create curve from the position of the particle as heat transfer coefficients.
		for (int i = 0; i < (referenceCooldownCurveSizeRows[0] * 2); i += 2)
		{
			double heatConductivity = 0;
			double specificHeat = 0;
			double heatTransferCoefficient = 0;
			double temperature = currentTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)];

			if (heatConductivitiesSizeRows[0] > 0)
			{
				double heatConductivity0 = 0;
				double heatConductivity1 = 0;
				double temperature0 = 0;
				double temperature1 = 0;
				int iterator = 0;

				while ((iterator < (heatConductivitiesSizeRows[0] * 2)) &&
					(heatConductivities[iterator + 0] <= temperature))
				{
					heatConductivity0 = heatConductivities[iterator + 1];
					temperature0 = heatConductivities[iterator + 0];
					iterator += 2;
				}

				if (iterator < (heatConductivitiesSizeRows[0] * 2))
				{
					heatConductivity1 = heatConductivities[iterator + 1];
					temperature1 = heatConductivities[iterator + 0];

					if (iterator == 0)
					{
						temperature0 = 0;
						heatConductivity0 = 0;
					}

					heatConductivity = ((heatConductivity1 - heatConductivity0) / (temperature1 - temperature0) * temperature -
						((heatConductivity1 - heatConductivity0) / (temperature1 - temperature0) * temperature0 - heatConductivity0));
				}
				else
				{
					heatConductivity = heatConductivities[(heatConductivitiesSizeRows[0] * 2) - 1];
				}
			}

			if (specificHeatsSizeRows[0] > 0)
			{
				double specificHeat0 = 0;
				double specificHeat1 = 0;
				double temperature0 = 0;
				double temperature1 = 0;
				int iterator = 0;

				while ((iterator < (specificHeatsSizeRows[0] * 2)) &&
					(specificHeats[iterator + 0] <= temperature))
				{
					specificHeat0 = specificHeats[iterator + 1];
					temperature0 = specificHeats[iterator + 0];
					iterator += 2;
				}

				if (iterator < (specificHeatsSizeRows[0] * 2))
				{
					specificHeat1 = specificHeats[iterator + 1];
					temperature1 = specificHeats[iterator + 0];

					if (iterator == 0)
					{
						temperature0 = 0;
						specificHeat0 = 0;
					}

					specificHeat = ((specificHeat1 - specificHeat0) / (temperature1 - temperature0) * temperature -
						((specificHeat1 - specificHeat0) / (temperature1 - temperature0) * temperature0 - specificHeat0));
				}
				else
				{
					specificHeat = specificHeats[(specificHeatsSizeRows[0] * 2) - 1];
				}
			}

			double alpha = (heatConductivity / (specificHeat * density[0]));
			currentTemperature[(index * horizontalSplitting[0]) + 0] = previousTemperature[(index * horizontalSplitting[0]) + 0] + timeDifference[0] * alpha * (1 / (dX[0] * dX[0]) * 2 *
				(previousTemperature[(index * horizontalSplitting[0]) + 1] - previousTemperature[(index * horizontalSplitting[0]) + 0]) + g[(index * horizontalSplitting[0]) + 0] / heatConductivity);

			if (dimensionNumber[0] > 0)
			{
				double heatTransferCoefficient0 = 0;
				double heatTransferCoefficient1 = 0;
				double temperature0 = 0;
				double temperature1 = 0;
				int iterator = 0;

				while ((iterator < (dimensionNumber[0] * 2)) &&
					(htcValues[iterator + 0] <= temperature))
				{
					heatTransferCoefficient0 = particlePosition[(index * dimensionNumber[0]) + (iterator / 2)];
					temperature0 = htcValues[iterator + 0];
					iterator += 2;
				}

				if (iterator < (dimensionNumber[0] * 2))
				{
					heatTransferCoefficient1 = particlePosition[(index * dimensionNumber[0]) + (iterator / 2)];
					temperature1 = htcValues[iterator + 0];

					if (iterator == 0)
					{
						temperature0 = 0;
						heatTransferCoefficient0 = 0;
					}

					heatTransferCoefficient = ((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature -
						((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature0 - heatTransferCoefficient0));
				}
				else
				{
					heatTransferCoefficient = particlePosition[(index * dimensionNumber[0]) + (dimensionNumber[0] - 1)];
				}
			}

			currentTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] = previousTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] +
				timeDifference[0] * alpha * (1 / (dX[0] * dX[0]) * 2 * (previousTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 2)] -
					previousTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] - dX[0] / heatConductivity * (heatTransferCoefficient *
					(previousTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] - tK[0]))) + 1 / (horizontalSplitting[0] * dX[0]) *
						(-1 / heatConductivity) * (heatTransferCoefficient * (previousTemperature[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] -
							tK[0])) + g[(index * horizontalSplitting[0]) + (horizontalSplitting[0] - 1)] / heatConductivity);

			for (int j = 1; j < (horizontalSplitting[0] - 1); j += 1)
			{
				currentTemperature[(index * horizontalSplitting[0]) + j] = previousTemperature[(index * horizontalSplitting[0]) + j] + timeDifference[0] * alpha * (1 / (dX[0] * dX[0]) *
					(previousTemperature[(index * horizontalSplitting[0]) + (j - 1)] + previousTemperature[(index * horizontalSplitting[0]) + (j + 1)] - 2 *
						previousTemperature[(index * horizontalSplitting[0]) + j]) + 1 / (j * dX[0]) * 1 / (2 * dX[0]) * (previousTemperature[(index * horizontalSplitting[0]) + (j + 1)] -
							previousTemperature[(index * horizontalSplitting[0]) + (j - 1)]) + g[(index * horizontalSplitting[0]) + j] / heatConductivity);
			}

			particleFitness[index] += (currentTemperature[(index * horizontalSplitting[0]) + monitoredIndex[0]] - referenceCooldownCurve[i + 1]) * (currentTemperature[(index * horizontalSplitting[0]) + monitoredIndex[0]] - referenceCooldownCurve[i + 1]);

			// Set the current temperature values as the previous temperature values for the next iteration.
			for (int j = 0; j < horizontalSplitting[0]; j += 1)
			{
				previousTemperature[(index * horizontalSplitting[0]) + j] = currentTemperature[(index * horizontalSplitting[0]) + j];
			}
		}

		if (particleFitness[index] < particleBestFitness[index])
		{
			particleBestFitness[index] = particleFitness[index];

			for (int j = 0; j < dimensionNumber[0]; j += 1)
			{
				particleBestPosition[(index * dimensionNumber[0]) + j] = particlePosition[(index * dimensionNumber[0]) + j];
			}
		}
	}
}

int main()
{
	hipSetDevice(0);
	ReadDataFromFile("ConfigurationIn.txt");
	CalculateReferenceCooldownCurve();
	WriteReferenceCooldownLogToFile();
	// Set particle swarm initial values.
	InitializeParticle();

	if (_cudaStatus != hipSuccess)
	{
		WriteCudaErrorToFile();
		hipDeviceReset();

		return -1;
	}

	//WritePreIterationResult();
	// Set informers.
	UpdateRing();

	// Set initial best values.
	for (int i = 0; i < _particleSwarmSize[0]; i += 1)
	{
		if (_particleBestFitness[i] < _globalBestFitness[_globalBestSize[0] - 1])
		{
			_globalBestFitness[_globalBestSize[0] - 1] = _particleBestFitness[i];

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				_globalBestPosition[(_globalBestSize[0] - 1) + j] = _particlePosition[(i * _dimensionNumber[0]) + j];
			}
		}
	}

	clock_t start = clock();
	Optimize();
	clock_t finish = clock();

	if (_cudaStatus != hipSuccess)
	{
		WriteCudaErrorToFile();
		hipDeviceReset();

		return -1;
	}

	WritePsoGlobalBestLogToFile();
	WriteExitResultAndTimeLogToFile((long)(finish - start));
	hipDeviceReset();

	return 0;
}

void InitializeParticle()
{
#pragma region InitializeParticle

	//hipDeviceReset();
	//hipSetDevice(0);
	double* _randomNumbers1 = new double[_dimensionNumber[0] * _particleSwarmSize[0]];
	double* _randomNumbers2 = new double[_dimensionNumber[0] * _particleSwarmSize[0]];

	for (int i = 0; i < (_dimensionNumber[0] * _particleSwarmSize[0]); i += 1)
	{
		_randomNumbers1[i] = ((double)rand() / RAND_MAX);
		_randomNumbers2[i] = ((double)rand() / RAND_MAX);
	}

	// Create device variables.
	int* dimensionNumber1;
	double* particlePosition1;
	double* particleVelocity1;
	double* particleInitialPosition;
	int* particleInitialPositionSize;
	double* particleInitialVelocity;
	int* particleInitialVelocitySize;
	int* rangeMin;
	int* rangeMax;
	double* randomNumbers1;
	double* randomNumbers2;

	// Allocate device memory for variables.
	hipMalloc((void**)&dimensionNumber1, sizeof(int));
	hipMalloc((void**)&particlePosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particleVelocity1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particleInitialPosition, _particleInitialPositionSize[0] * sizeof(double));
	hipMalloc((void**)&particleInitialPositionSize, sizeof(int));
	hipMalloc((void**)&particleInitialVelocity, _particleInitialVelocitySize[0] * sizeof(double));
	hipMalloc((void**)&particleInitialVelocitySize, sizeof(int));
	hipMalloc((void**)&rangeMin, sizeof(int));
	hipMalloc((void**)&rangeMax, sizeof(int));
	hipMalloc((void**)&randomNumbers1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&randomNumbers2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));

	// Copy host variable values to device variables.
	hipMemcpy(dimensionNumber1, _dimensionNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particlePosition1, _particlePosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleVelocity1, _particleVelocity, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleInitialPosition, _particleInitialPosition, _particleInitialPositionSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleInitialPositionSize, _particleInitialPositionSize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleInitialVelocity, _particleInitialVelocity, _particleInitialVelocitySize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleInitialVelocitySize, _particleInitialVelocitySize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rangeMin, _rangeMin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rangeMax, _rangeMax, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(randomNumbers1, _randomNumbers1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(randomNumbers2, _randomNumbers2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);

	// Free memory.
	delete[] _randomNumbers1;
	delete[] _randomNumbers2;

	// Call kernel function.
	InitializeParticleKernel << <1, _particleSwarmSize[0] >> >(dimensionNumber1,
		particlePosition1, particleVelocity1, particleInitialPosition,
		particleInitialPositionSize, particleInitialVelocity,
		particleInitialVelocitySize, rangeMin, rangeMax, randomNumbers1,
		randomNumbers2);

	_cudaStatus = hipGetLastError();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << hipGetErrorString(_cudaStatus);
		_cudaError += "InitializeParticle launch failed: " + ss.str() + "\n";
		fprintf(stderr, "InitializeParticle launch failed: %s\n", hipGetErrorString(_cudaStatus));
	}
	// Wait for all threads to finish.
	_cudaStatus = hipDeviceSynchronize();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << _cudaStatus;
		_cudaError += "hipDeviceSynchronize returned error code " + ss.str() + " after launching InitializeParticleKernel!\n";
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching InitializeParticleKernel!\n", _cudaStatus);
	}

	// Copy device variable values to host variables.
	hipMemcpy(_particlePosition, particlePosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleVelocity, particleVelocity1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);

	// Free device variables.
	hipFree(dimensionNumber1);
	hipFree(particlePosition1);
	hipFree(particleVelocity1);
	hipFree(particleInitialPosition);
	hipFree(particleInitialPositionSize);
	hipFree(particleInitialVelocity);
	hipFree(particleInitialVelocitySize);
	hipFree(rangeMin);
	hipFree(rangeMax);
	hipFree(randomNumbers1);
	hipFree(randomNumbers2);
	//hipDeviceReset();

#pragma endregion

#pragma region ObjectiveFunction

	//hipDeviceReset();
	//hipSetDevice(0);
	double* _currentTemperature = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];
	double* _previousTemperature = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];
	double* _g = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];

	for (int i = 0; i < (_horizontalSplitting[0] * _particleSwarmSize[0]); i += 1)
	{
		_currentTemperature[i] = _initialTemperature[0];
		_previousTemperature[i] = _initialTemperature[0];
		_g[i] = 0;
	}

	// Create device variables.
	int* dimensionNumber2;
	double* particleBestFitness2;
	double* particleBestPosition2;
	double* particlePosition2;
	int* rangeMin2;
	int* rangeMax2;
	double* particleFitness;
	double* initialTemperature;
	int* horizontalSplitting;
	int* referenceCooldownCurveSizeRows;
	double* referenceCooldownCurve;
	int* monitoredIndex;
	double* timeDifference;
	double* dX;
	double* tK;
	int* heatConductivitiesSizeRows;
	double* heatConductivities;
	int* specificHeatsSizeRows;
	double* specificHeats;
	double* density;
	double* htcValues;
	double* currentTemperature;
	double* previousTemperature;
	double* g;

	// Allocate device memory for variables.
	hipMalloc((void**)&dimensionNumber2, sizeof(int));
	hipMalloc((void**)&particleBestFitness2, _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&particleBestPosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particlePosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&rangeMin2, sizeof(int));
	hipMalloc((void**)&rangeMax2, sizeof(int));
	hipMalloc((void**)&particleFitness, _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&initialTemperature, sizeof(double));
	hipMalloc((void**)&horizontalSplitting, sizeof(int));
	hipMalloc((void**)&referenceCooldownCurveSizeRows, sizeof(int));
	hipMalloc((void**)&referenceCooldownCurve, _referenceCooldownCurveSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&monitoredIndex, sizeof(int));
	hipMalloc((void**)&timeDifference, sizeof(double));
	hipMalloc((void**)&dX, sizeof(double));
	hipMalloc((void**)&tK, sizeof(double));
	hipMalloc((void**)&heatConductivitiesSizeRows, sizeof(int));
	hipMalloc((void**)&heatConductivities, _heatConductivitiesSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&specificHeatsSizeRows, sizeof(int));
	hipMalloc((void**)&specificHeats, _specificHeatsSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&density, sizeof(double));
	hipMalloc((void**)&htcValues, _dimensionNumber[0] * 2 * sizeof(double));
	hipMalloc((void**)&currentTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&previousTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&g, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));

	// Copy host variable values to device variables.
	hipMemcpy(dimensionNumber2, _dimensionNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleBestFitness2, _particleBestFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleBestPosition2, _particleBestPosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particlePosition2, _particlePosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(rangeMin2, _rangeMin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rangeMax2, _rangeMax, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleFitness, _particleFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(initialTemperature, _initialTemperature, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(horizontalSplitting, _horizontalSplitting, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(referenceCooldownCurveSizeRows, _referenceCooldownCurveSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(referenceCooldownCurve, _referenceCooldownCurve, _referenceCooldownCurveSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(monitoredIndex, _monitoredIndex, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(timeDifference, _timeDifference, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dX, _dX, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(tK, _tK, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(heatConductivitiesSizeRows, _heatConductivitiesSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(heatConductivities, _heatConductivities, _heatConductivitiesSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(specificHeatsSizeRows, _specificHeatsSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(specificHeats, _specificHeats, _specificHeatsSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(density, _density, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(htcValues, _htcValues, _dimensionNumber[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(currentTemperature, _currentTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(previousTemperature, _previousTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(g, _g, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);

	// Free memory.
	delete[] _currentTemperature;
	delete[] _previousTemperature;
	delete[] _g;

	// Call kernel function.
	ObjectiveFunctionKernel << <1, _particleSwarmSize[0] >> > (dimensionNumber2,
		particleBestFitness2, particleBestPosition2, particlePosition2,
		rangeMin2, rangeMax2, particleFitness, initialTemperature, horizontalSplitting,
		referenceCooldownCurveSizeRows, referenceCooldownCurve, monitoredIndex,
		timeDifference, dX, tK, heatConductivitiesSizeRows, heatConductivities,
		specificHeatsSizeRows, specificHeats, density, htcValues,
		currentTemperature, previousTemperature, g);

	_cudaStatus = hipGetLastError();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << hipGetErrorString(_cudaStatus);
		_cudaError += "ObjectiveFunctionKernel launch failed: " + ss.str() + "\n";
		fprintf(stderr, "ObjectiveFunctionKernel launch failed: %s\n", hipGetErrorString(_cudaStatus));
	}
	// Wait for all threads to finish.
	_cudaStatus = hipDeviceSynchronize();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << _cudaStatus;
		_cudaError += "hipDeviceSynchronize returned error code " + ss.str() + " after launching ObjectiveFunctionKernel!\n";
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching ObjectiveFunctionKernel!\n", _cudaStatus);
	}

	// Copy device variable values to host variables.
	hipMemcpy(_particleBestFitness, particleBestFitness2, _particleSwarmSize[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleBestPosition, particleBestPosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particlePosition, particlePosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleFitness, particleFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyDeviceToHost);

	// Free device variables.
	hipFree(dimensionNumber2);
	hipFree(particleBestFitness2);
	hipFree(particleBestPosition2);
	hipFree(particlePosition2);
	hipFree(rangeMin2);
	hipFree(rangeMax2);
	hipFree(particleFitness);
	hipFree(initialTemperature);
	hipFree(horizontalSplitting);
	hipFree(referenceCooldownCurveSizeRows);
	hipFree(referenceCooldownCurve);
	hipFree(monitoredIndex);
	hipFree(timeDifference);
	hipFree(dX);
	hipFree(tK);
	hipFree(heatConductivitiesSizeRows);
	hipFree(heatConductivities);
	hipFree(specificHeatsSizeRows);
	hipFree(specificHeats);
	hipFree(density);
	hipFree(htcValues);
	hipFree(currentTemperature);
	hipFree(previousTemperature);
	hipFree(g);
	//hipDeviceReset();

	for (int i = 0; i < _particleSwarmSize[0]; i += 1)
	{
		_particleBestFitness[i] = _particleFitness[i];

		for (int j = 0; j < _dimensionNumber[0]; j += 1)
		{
			_particleBestPosition[(i * _dimensionNumber[0]) + j] = _particlePosition[(i * _dimensionNumber[0]) + j];
		}
	}

#pragma endregion
}

void ReadDataFromFile(std::string dataFilePath)
{
	std::ifstream fileStream(dataFilePath);

	if (fileStream.is_open())
	{
		std::string currentLine = "";

		while (std::getline(fileStream, currentLine))
		{
			if (currentLine.find("#") != std::string::npos)
			{
				std::transform(currentLine.begin(), currentLine.end(), currentLine.begin(), ::tolower);

				if (currentLine.find("initial temperature") != std::string::npos)
				{
					std::getline(fileStream, currentLine);
					std::remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_initialTemperature = new double[1];
					_initialTemperature[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("horizontal splitting") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_horizontalSplitting = new int[1];
					_horizontalSplitting[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("vertical splitting") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_verticalSplitting = new int[1];
					_verticalSplitting[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("monitored index") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_monitoredIndex = new int[1];
					_monitoredIndex[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("range min") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_rangeMin = new int[1];
					_rangeMin[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("range max") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_rangeMax = new int[1];
					_rangeMax[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("simulation time") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_simulationTime = new double[1];
					_simulationTime[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("height") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_height = new double[1];
					_height[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("htcin") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					dataFilePath = currentLine;
				}
				else if (currentLine.find("time difference") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_timeDifference = new double[1];
					_timeDifference[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("pso epsilon") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleEpsilon = new double[1];
					_particleEpsilon[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("radius") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_radius = new double[1];
					_radius[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("density") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_density = new double[1];
					_density[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("optimization type") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					_particleOptimalisationType = new int[1];

					if (currentLine == "clerc")
					{
						_particleOptimalisationType[0] = 1;
					}
					else if (currentLine == "quantum")
					{
						_particleOptimalisationType[0] = 2;
					}
					else
					{
						_particleOptimalisationType[0] = 1;
					}
				}
				else if (currentLine.find("swarm size") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleSwarmSize = new int[1];
					_particleSwarmSize[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("max epochs") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_maxEpochs = new int[1];
					_maxEpochs[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("max static epochs") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_maxStaticEpochs = new int[1];
					_maxStaticEpochs[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("weight") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_weight = new double[1];
					_weight[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("constant1") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleConstant1 = new double[1];
					_particleConstant1[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("constant2") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleConstant2 = new double[1];
					_particleConstant2[0] = atof(currentLine.c_str());
				}
				else if (currentLine.find("informer number") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleInformerNumber = new int[1];
					_particleInformerNumber[0] = atoi(currentLine.c_str());
				}
				else if (currentLine.find("initial position") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleInitialPositionSize = new int[1];
					_particleInitialPositionSize[0] = atoi(currentLine.c_str());

					if (_particleInitialPositionSize[0] > 0)
					{
						_particleInitialPosition = new double[_particleInitialPositionSize[0]];

						for (int i = 0; i < _particleInitialPositionSize[0]; i += 1)
						{
							std::getline(fileStream, currentLine);
							remove(currentLine.begin(), currentLine.end(), ' ');
							replace(currentLine.begin(), currentLine.end(), ',', '.');
							_particleInitialPosition[i] = atof(currentLine.c_str());
						}
					}
				}
				else if (currentLine.find("initial velocity") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_particleInitialVelocitySize = new int[1];
					_particleInitialVelocitySize[0] = atoi(currentLine.c_str());

					if (_particleInitialVelocitySize[0] > 0)
					{
						_particleInitialVelocity = new double[_particleInitialVelocitySize[0]];

						for (int i = 0; i < _particleInitialVelocitySize[0]; i += 1)
						{
							std::getline(fileStream, currentLine);
							remove(currentLine.begin(), currentLine.end(), ' ');
							replace(currentLine.begin(), currentLine.end(), ',', '.');
							_particleInitialVelocity[i] = atof(currentLine.c_str());
						}
					}
				}
				else if (currentLine.find("heat conductivity") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_heatConductivitiesSizeRows = new int[1];
					_heatConductivitiesSizeRows[0] = atoi(currentLine.c_str());

					if (_heatConductivitiesSizeRows[0] > 0)
					{
						_heatConductivities = new double[_heatConductivitiesSizeRows[0] * 2];

						for (int i = 0; i < (_heatConductivitiesSizeRows[0] * 2); i += 2)
						{
							std::getline(fileStream, currentLine);
							replace(currentLine.begin(), currentLine.end(), ',', '.');
							replace(currentLine.begin(), currentLine.end(), '\t', ' ');
							std::vector<std::string> result = explode(currentLine, ' ');
							_heatConductivities[i + 0] = atof(result[0].c_str());
							_heatConductivities[i + 1] = atof(result[1].c_str());
						}
					}
				}
				else if (currentLine.find("specific heat") != std::string::npos)
				{
					getline(fileStream, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_specificHeatsSizeRows = new int[1];
					_specificHeatsSizeRows[0] = atoi(currentLine.c_str());

					if (_specificHeatsSizeRows[0] > 0)
					{
						_specificHeats = new double[_specificHeatsSizeRows[0] * 2];

						for (int i = 0; i < (_specificHeatsSizeRows[0] * 2); i += 2)
						{
							std::getline(fileStream, currentLine);
							replace(currentLine.begin(), currentLine.end(), ',', '.');
							replace(currentLine.begin(), currentLine.end(), '\t', ' ');
							std::vector<std::string> result = explode(currentLine, ' ');
							_specificHeats[i + 0] = atof(result[0].c_str());
							_specificHeats[i + 1] = atof(result[1].c_str());
						}
					}
				}
			}
		}

		fileStream.close();
	}

	std::ifstream fileStream2(dataFilePath);

	if (fileStream2.is_open())
	{
		std::string currentLine = "";

		while (std::getline(fileStream2, currentLine))
		{
			if (currentLine.find("#") != std::string::npos)
			{
				std::transform(currentLine.begin(), currentLine.end(), currentLine.begin(), ::tolower);

				if (currentLine.find("htc values") != std::string::npos)
				{
					std::getline(fileStream2, currentLine);
					remove(currentLine.begin(), currentLine.end(), ' ');
					replace(currentLine.begin(), currentLine.end(), ',', '.');
					_dimensionNumber = new int[1];
					_dimensionNumber[0] = atoi(currentLine.c_str());

					if (_dimensionNumber[0] > 0)
					{
						_htcValues = new double[_dimensionNumber[0] * 2];

						for (int i = 0; i < (_dimensionNumber[0] * 2); i += 2)
						{
							std::getline(fileStream2, currentLine);
							replace(currentLine.begin(), currentLine.end(), ',', '.');
							std::vector<std::string> result = explode(currentLine, ' ');
							_htcValues[i + 0] = atof(result[0].c_str());
							_htcValues[i + 1] = atof(result[1].c_str());
						}
					}
				}
			}
		}

		fileStream2.close();
	}

	_dX = new double[1];
	_dX[0] = (_radius[0] / 1000) / _horizontalSplitting[0];
	_dY = new double[1];
	_dY[0] = (_height[0] / 1000) / _verticalSplitting[0];
	_tK = new double[1];
	_tK[0] = 0;
	_cudaStatus = hipSuccess;
	_exitReason = "";
	// Get particle swarm initial values.
	_particlePosition = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	_particleVelocity = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	_particleBestPosition = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	_particleFitness = new double[_particleSwarmSize[0]];
	_particleBestFitness = new double[_particleSwarmSize[0]];
	_globalBestSize = new int[1];
	_globalBestSize[0] = 1;
	_globalBestFitness = new double[_globalBestSize[0]];
	_globalBestFitness[_globalBestSize[0] - 1] = DBL_MAX;
	_globalBestPosition = new double[_globalBestSize[0] * _dimensionNumber[0]];
	_epoch = new int[1];
	_epoch[0] = 0;
}

void CalculateReferenceCooldownCurve()
{
	_referenceCooldownCurveSizeRows = new int[1];
	_referenceCooldownCurveSizeRows[0] = (int)(_simulationTime[0] / _timeDifference[0]) + 1;
	_referenceCooldownCurve = new double[_referenceCooldownCurveSizeRows[0] * 2];
	double* currentTemperature = new double[_horizontalSplitting[0]];
	double* previousTemperature = new double[_horizontalSplitting[0]];
	double* g = new double[_horizontalSplitting[0]];

	for (int i = 0; i < _horizontalSplitting[0]; i += 1)
	{
		currentTemperature[i] = _initialTemperature[0];
		previousTemperature[i] = _initialTemperature[0];
		g[i] = 0;
	}

	for (int i = 0; i < (_referenceCooldownCurveSizeRows[0] * 2); i += 2)
	{
		currentTemperature = CalculateCooldownCurve1D(false, -1, currentTemperature, previousTemperature, g);
		_referenceCooldownCurve[i + 0] = ((i / 2) * _timeDifference[0]);
		_referenceCooldownCurve[i + 1] = currentTemperature[_monitoredIndex[0]];

		for (int j = 0; j < _horizontalSplitting[0]; j += 1)
		{
			previousTemperature[j] = currentTemperature[j];
		}
	}

	delete[] currentTemperature;
	delete[] previousTemperature;
	delete[] g;
}

double* CalculateCooldownCurve1D(bool isParticle, int particleIndex,
	double* currentTemperature, double* previousTemperature, double* g)
{
	double temperature = currentTemperature[_horizontalSplitting[0] - 1];
	double heatConductivity = GetHeatConductivity(temperature);
	double alpha = GetAlpha(heatConductivity, GetSpecificHeat(temperature));
	currentTemperature[0] = previousTemperature[0] + _timeDifference[0] * alpha * (1 / (_dX[0] * _dX[0]) * 2 *
		(previousTemperature[1] - previousTemperature[0]) + g[0] / heatConductivity);
	double heatTransferCoefficient = GetHeatTransferCoefficient(temperature, isParticle, particleIndex);
	currentTemperature[_horizontalSplitting[0] - 1] = previousTemperature[_horizontalSplitting[0] - 1] +
		_timeDifference[0] * alpha * (1 / (_dX[0] * _dX[0]) * 2 * (previousTemperature[_horizontalSplitting[0] - 2] -
			previousTemperature[_horizontalSplitting[0] - 1] - _dX[0] / heatConductivity * (heatTransferCoefficient *
			(previousTemperature[_horizontalSplitting[0] - 1] - _tK[0]))) + 1 / (_horizontalSplitting[0] * _dX[0]) *
				(-1 / heatConductivity) * (heatTransferCoefficient * (previousTemperature[_horizontalSplitting[0] - 1] -
					_tK[0])) + g[_horizontalSplitting[0] - 1] / heatConductivity);

	for (int i = 1; i < _horizontalSplitting[0] - 1; i += 1)
	{
		currentTemperature[i] = previousTemperature[i] + _timeDifference[0] * alpha * (1 / (_dX[0] * _dX[0]) *
			(previousTemperature[i - 1] + previousTemperature[i + 1] - 2 * previousTemperature[i]) +
			1 / (i * _dX[0]) * 1 / (2 * _dX[0]) * (previousTemperature[i + 1] - previousTemperature[i - 1]) +
			g[i] / heatConductivity);
	}

	return currentTemperature;
}

double GetHeatConductivity(double temperature)
{
	if (_heatConductivitiesSizeRows[0] > 0)
	{
		double heatConductivity0 = 0;
		double heatConductivity1 = 0;
		double temperature0 = 0;
		double temperature1 = 0;
		int i = 0;

		while ((i < (_heatConductivitiesSizeRows[0] * 2)) &&
			(_heatConductivities[i + 0] <= temperature))
		{
			heatConductivity0 = _heatConductivities[i + 1];
			temperature0 = _heatConductivities[i + 0];
			i += 2;
		}

		if (i < (_heatConductivitiesSizeRows[0] * 2))
		{
			heatConductivity1 = _heatConductivities[i + 1];
			temperature1 = _heatConductivities[i + 0];

			if (i == 0)
			{
				temperature0 = 0;
				heatConductivity0 = 0;
			}

			return ((heatConductivity1 - heatConductivity0) / (temperature1 - temperature0) * temperature -
				((heatConductivity1 - heatConductivity0) / (temperature1 - temperature0) * temperature0 - heatConductivity0));
		}
		else
		{
			return _heatConductivities[(_heatConductivitiesSizeRows[0] * 2) - 1];
		}
	}

	return 0;
}

double GetSpecificHeat(double temperature)
{
	if (_specificHeatsSizeRows[0] > 0)
	{
		double specificHeat0 = 0;
		double specificHeat1 = 0;
		double temperature0 = 0;
		double temperature1 = 0;
		int i = 0;

		while ((i < (_specificHeatsSizeRows[0] * 2)) &&
			(_specificHeats[i + 0] <= temperature))
		{
			specificHeat0 = _specificHeats[i + 1];
			temperature0 = _specificHeats[i + 0];
			i += 2;
		}

		if (i < (_specificHeatsSizeRows[0] * 2))
		{
			specificHeat1 = _specificHeats[i + 1];
			temperature1 = _specificHeats[i + 0];

			if (i == 0)
			{
				temperature0 = 0;
				specificHeat0 = 0;
			}

			return ((specificHeat1 - specificHeat0) / (temperature1 - temperature0) * temperature -
				((specificHeat1 - specificHeat0) / (temperature1 - temperature0) * temperature0 - specificHeat0));
		}
		else
		{
			return _specificHeats[(_specificHeatsSizeRows[0] * 2) - 1];
		}
	}

	return 0;
}

double GetAlpha(double heatConductivity, double specificHeat)
{
	return (heatConductivity / (specificHeat * _density[0]));
}

double GetHeatTransferCoefficient(double temperature, bool isParticle, int particleIndex)
{
	if (!isParticle)
	{
		if (_dimensionNumber[0] > 0)
		{
			double heatTransferCoefficient0 = 0;
			double heatTransferCoefficient1 = 0;
			double temperature0 = 0;
			double temperature1 = 0;
			int i = 0;

			while ((i < (_dimensionNumber[0] * 2)) &&
				(_htcValues[i + 0] <= temperature))
			{
				heatTransferCoefficient0 = _htcValues[i + 1];
				temperature0 = _htcValues[i + 0];
				i += 2;
			}

			if (i < (_dimensionNumber[0] * 2))
			{
				heatTransferCoefficient1 = _htcValues[i + 1];
				temperature1 = _htcValues[i + 0];

				if (i == 0)
				{
					temperature0 = 0;
					heatTransferCoefficient0 = 0;
				}

				return ((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature -
					((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature0 - heatTransferCoefficient0));
			}
			else
			{
				return _htcValues[(_dimensionNumber[0] * 2) - 1];
			}
		}

		return 0;
	}
	else
	{
		if (_dimensionNumber[0] > 0)
		{
			double heatTransferCoefficient0 = 0;
			double heatTransferCoefficient1 = 0;
			double temperature0 = 0;
			double temperature1 = 0;
			int i = 0;

			while ((i < (_dimensionNumber[0] * 2)) &&
				(_htcValues[i + 0] <= temperature))
			{
				heatTransferCoefficient0 = _particlePosition[(particleIndex * _dimensionNumber[0]) + (i / 2)];
				temperature0 = _htcValues[i + 0];
				i += 2;
			}

			if (i < (_dimensionNumber[0] * 2))
			{
				heatTransferCoefficient1 = _particlePosition[(particleIndex * _dimensionNumber[0]) + (i / 2)];
				temperature1 = _htcValues[i + 0];

				if (i == 0)
				{
					temperature0 = 0;
					heatTransferCoefficient0 = 0;
				}

				return ((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature -
					((heatTransferCoefficient1 - heatTransferCoefficient0) / (temperature1 - temperature0) * temperature0 - heatTransferCoefficient0));
			}
			else
			{
				return _particlePosition[(particleIndex * _dimensionNumber[0]) + (_dimensionNumber[0] - 1)];
			}
		}

		return 0;
	}
}

void WriteReferenceCooldownLogToFile()
{
	if (_referenceCooldownCurve != NULL && _referenceCooldownCurveSizeRows[0] > 0)
	{
		std::ofstream fileStream("ReferenceCooldownLog.txt");

		if (fileStream.is_open())
		{
			fileStream << "time ";
			fileStream << "temperature ";
			fileStream << "\r\n";

			for (int i = 0; i < (_referenceCooldownCurveSizeRows[0] * 2); i += 2)
			{
				std::stringstream ss1;
				ss1 << std::fixed << std::setprecision(3) << _referenceCooldownCurve[i + 0];
				std::string timeData = ss1.str();
				std::replace(timeData.begin(), timeData.end(), ',', '.');
				std::stringstream ss2;
				ss2 << std::fixed << std::setprecision(3) << _referenceCooldownCurve[i + 1];
				std::string temperatureData = ss2.str();
				replace(temperatureData.begin(), temperatureData.end(), ',', '.');
				fileStream << timeData << " ";
				fileStream << temperatureData << " ";
				fileStream << "\r\n";
			}

			fileStream.close();
		}
	}
}

void WritePsoGlobalBestLogToFile()
{
	if (_globalBestFitness != NULL && _globalBestPosition != NULL &&
		_globalBestSize[0] > 0)
	{
		std::ofstream fileStream("ParticleSwarmOptimizationGlobalBestLog.txt");

		if (fileStream.is_open())
		{
			fileStream << "fitness ";

			for (int i = 0; i < _dimensionNumber[0]; i += 1)
			{
				std::stringstream ss;
				ss << (i + 1);
				fileStream << "htc" << ss.str() << " ";
			}

			fileStream << "\r\n";

			for (int i = 0; i < _globalBestSize[0]; i += 1)
			{
				std::stringstream ss1;
				ss1 << std::fixed << std::setprecision(3) << _globalBestFitness[i];
				std::string value = ss1.str();
				std::replace(value.begin(), value.end(), ',', '.');
				fileStream << value << " ";

				for (int j = 0; j < _dimensionNumber[0]; j += 1)
				{
					std::stringstream ss2;
					ss2 << std::fixed << std::setprecision(3) << _globalBestPosition[(i * _dimensionNumber[0]) + j];
					value = ss2.str();
					replace(value.begin(), value.end(), ',', '.');
					fileStream << value << " ";
				}

				fileStream << "\r\n";
			}

			fileStream.close();
		}
	}
}

void WriteExitResultAndTimeLogToFile(long elaspedMilliseconds)
{
	std::ofstream fileStream("ExitResultAndTimeLog.txt");

	if (fileStream.is_open())
	{
		std::stringstream ss;
		ss << elaspedMilliseconds;
		std::stringstream ss2;
		ss2 << _epoch[0];
		fileStream << "Exit reason: " << _exitReason << ", elapsed time: " << ss.str() << "ms, iterations: " << ss2.str();
		fileStream.close();
	}
}

void WriteCudaErrorToFile()
{
	std::ofstream fileStream("CudaError.txt");

	if (fileStream.is_open())
	{
		fileStream << _cudaError;
		fileStream.close();
	}
}

void WritePreIterationResult()
{
	std::ofstream fileStream("PreIteration.txt");

	if (fileStream.is_open())
	{
		fileStream << "fitness ";

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "position" << ss.str() << " ";
		}

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "velocity" << ss.str() << " ";
		}

		fileStream << "bestFitness ";

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "bestPosition" << ss.str() << " ";
		}

		fileStream << "\r\n";

		for (int i = 0; i < _particleSwarmSize[0]; i += 1)
		{
			std::stringstream ss1;
			ss1 << std::fixed << std::setprecision(3) << _particleFitness[i];
			fileStream << ss1.str() << " ";

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particlePosition[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particleVelocity[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			std::stringstream ss2;
			ss2 << std::fixed << std::setprecision(3) << _particleBestFitness[i];
			fileStream << ss2.str() << " ";

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particleBestPosition[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			fileStream << "\r\n";
		}

		fileStream.close();
	}
}

void WriteIterationResult(int iteration)
{
	std::stringstream ss0;
	ss0 << iteration;
	std::ofstream fileStream("Iteration" + ss0.str() + ".txt");

	if (fileStream.is_open())
	{
		fileStream << "fitness ";

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "position" << ss.str() << " ";
		}

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "velocity" << ss.str() << " ";
		}

		fileStream << "bestFitness ";

		for (int i = 0; i < _dimensionNumber[0]; i += 1)
		{
			std::stringstream ss;
			ss << (i + 1);
			fileStream << "bestPosition" << ss.str() << " ";
		}

		fileStream << "\r\n";

		for (int i = 0; i < _particleSwarmSize[0]; i += 1)
		{
			std::stringstream ss1;
			ss1 << std::fixed << std::setprecision(3) << _particleFitness[i];
			fileStream << ss1.str() << " ";

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particlePosition[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particleVelocity[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			std::stringstream ss2;
			ss2 << std::fixed << std::setprecision(3) << _particleBestFitness[i];
			fileStream << ss2.str() << " ";

			for (int j = 0; j < _dimensionNumber[0]; j += 1)
			{
				std::stringstream ss;
				ss << std::fixed << std::setprecision(3) << _particleBestPosition[(i * _dimensionNumber[0]) + j];
				fileStream << ss.str() << " ";
			}

			fileStream << "\r\n";
		}

		fileStream.close();
	}
}

void OptimizePosition(double* beta, double* mBest)
{
#pragma region OptimizePosition

	//hipDeviceReset();
	//hipSetDevice(0);
	double* _randomNumbers1 = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	double* _randomNumbers2 = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	double* _randomNumbers3 = new double[_particleSwarmSize[0] * _dimensionNumber[0]];
	double* _bestLocalPosition = new double[_dimensionNumber[0]];

	for (int i = 0; i < (_particleSwarmSize[0] * _dimensionNumber[0]); i += 1)
	{
		_randomNumbers1[i] = ((double)rand() / RAND_MAX);
		_randomNumbers2[i] = ((double)rand() / RAND_MAX);
		_randomNumbers3[i] = ((double)rand() / RAND_MAX);
	}

	// Create device variables.
	int* particleOptimalisationType;
	double* dMBest;
	double* dBeta;
	int* dimensionNumber1;
	int* particleInformerNumber;
	int* particleInformers;
	double* particleBestFitness1;
	double* particleBestPosition1;
	double* weight;
	double* particleVelocity;
	double* particlePosition1;
	int* rangeMin1;
	int* rangeMax1;
	int* globalBestSize;
	double* globalBestPosition;
	double* particleConstant1;
	double* particleConstant2;
	double* randomNumbers1;
	double* randomNumbers2;
	double* randomNumbers3;
	double* bestLocalPosition;

	// Allocate device memory for variables.
	hipMalloc((void**)&particleOptimalisationType, sizeof(int));
	hipMalloc((void**)&dMBest, sizeof(double));
	hipMalloc((void**)&dBeta, sizeof(double));
	hipMalloc((void**)&dimensionNumber1, sizeof(int));
	hipMalloc((void**)&particleInformerNumber, sizeof(int));
	hipMalloc((void**)&particleInformers, _particleInformerNumber[0] * _particleSwarmSize[0] * sizeof(int));
	hipMalloc((void**)&particleBestFitness1, _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&particleBestPosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&weight, sizeof(double));
	hipMalloc((void**)&particleVelocity, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particlePosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&rangeMin1, sizeof(int));
	hipMalloc((void**)&rangeMax1, sizeof(int));
	hipMalloc((void**)&globalBestSize, sizeof(int));
	hipMalloc((void**)&globalBestPosition, _globalBestSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particleConstant1, sizeof(double));
	hipMalloc((void**)&particleConstant2, sizeof(double));
	hipMalloc((void**)&randomNumbers1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&randomNumbers2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&randomNumbers3, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&bestLocalPosition, _dimensionNumber[0] * sizeof(double));

	// Copy host variable values to device variables.
	hipMemcpy(particleOptimalisationType, _particleOptimalisationType, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dMBest, mBest, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dBeta, beta, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dimensionNumber1, _dimensionNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleInformerNumber, _particleInformerNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleInformers, _particleInformers, _particleInformerNumber[0] * _particleSwarmSize[0] * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleBestFitness1, _particleBestFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleBestPosition1, _particleBestPosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(weight, _weight, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleVelocity, _particleVelocity, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particlePosition1, _particlePosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(rangeMin1, _rangeMin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rangeMax1, _rangeMax, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(globalBestSize, _globalBestSize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(globalBestPosition, _globalBestPosition, _globalBestSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleConstant1, _particleConstant1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleConstant2, _particleConstant2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(randomNumbers1, _randomNumbers1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(randomNumbers2, _randomNumbers2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(randomNumbers3, _randomNumbers3, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(bestLocalPosition, _bestLocalPosition, _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);

	// Free memory.
	delete[] _randomNumbers1;
	delete[] _randomNumbers2;
	delete[] _randomNumbers3;
	delete[] _bestLocalPosition;

	// Call kernel function.
	OptimizePositionKernel << <1, _particleSwarmSize[0] >> >(particleOptimalisationType,
		dMBest, dBeta, dimensionNumber1, particleInformerNumber, particleInformers,
		particleBestFitness1, particleBestPosition1, weight, particleVelocity,
		particlePosition1, rangeMin1, rangeMax1, globalBestSize, globalBestPosition,
		particleConstant1, particleConstant2, randomNumbers1, randomNumbers2,
		randomNumbers3, bestLocalPosition);

	_cudaStatus = hipGetLastError();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << hipGetErrorString(_cudaStatus);
		_cudaError += "OptimizePosition launch failed: " + ss.str() + "\n";
		fprintf(stderr, "OptimizePosition launch failed: %s\n", hipGetErrorString(_cudaStatus));
	}
	// Wait for all threads to finish.
	hipDeviceSynchronize();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << _cudaStatus;
		_cudaError += "hipDeviceSynchronize returned error code " + ss.str() + " after launching OptimizePositionKernel!\n";
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching OptimizePositionKernel!\n", _cudaStatus);
	}

	// Copy device variable values to host variables.
	hipMemcpy(_particleBestFitness, particleBestFitness1, _particleSwarmSize[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleBestPosition, particleBestPosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleVelocity, particleVelocity, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particlePosition, particlePosition1, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);

	// Free device variables.
	hipFree(particleOptimalisationType);
	hipFree(dMBest);
	hipFree(dBeta);
	hipFree(dimensionNumber1);
	hipFree(particleInformerNumber);
	hipFree(particleInformers);
	hipFree(particleBestFitness1);
	hipFree(particleBestPosition1);
	hipFree(weight);
	hipFree(particleVelocity);
	hipFree(particlePosition1);
	hipFree(rangeMin1);
	hipFree(rangeMax1);
	hipFree(globalBestSize);
	hipFree(globalBestPosition);
	hipFree(particleConstant1);
	hipFree(particleConstant2);
	hipFree(randomNumbers1);
	hipFree(randomNumbers2);
	hipFree(randomNumbers3);
	hipFree(bestLocalPosition);
	//hipDeviceReset();

#pragma endregion

#pragma region ObjectiveFunction

	//hipDeviceReset();
	//hipSetDevice(0);
	double* _currentTemperature = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];
	double* _previousTemperature = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];
	double* _g = new double[_horizontalSplitting[0] * _particleSwarmSize[0]];

	for (int i = 0; i < (_horizontalSplitting[0] * _particleSwarmSize[0]); i += 1)
	{
		_currentTemperature[i] = _initialTemperature[0];
		_previousTemperature[i] = _initialTemperature[0];
		_g[i] = 0;
	}

	// Create device variables.
	int* dimensionNumber2;
	double* particleBestFitness2;
	double* particleBestPosition2;
	double* particlePosition2;
	int* rangeMin2;
	int* rangeMax2;
	double* particleFitness;
	double* initialTemperature;
	int* horizontalSplitting;
	int* referenceCooldownCurveSizeRows;
	double* referenceCooldownCurve;
	int* monitoredIndex;
	double* timeDifference;
	double* dX;
	double* tK;
	int* heatConductivitiesSizeRows;
	double* heatConductivities;
	int* specificHeatsSizeRows;
	double* specificHeats;
	double* density;
	double* htcValues;
	double* currentTemperature;
	double* previousTemperature;
	double* g;

	// Allocate device memory for variables.
	hipMalloc((void**)&dimensionNumber2, sizeof(int));
	hipMalloc((void**)&particleBestFitness2, _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&particleBestPosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&particlePosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double));
	hipMalloc((void**)&rangeMin2, sizeof(int));
	hipMalloc((void**)&rangeMax2, sizeof(int));
	hipMalloc((void**)&particleFitness, _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&initialTemperature, sizeof(double));
	hipMalloc((void**)&horizontalSplitting, sizeof(int));
	hipMalloc((void**)&referenceCooldownCurveSizeRows, sizeof(int));
	hipMalloc((void**)&referenceCooldownCurve, _referenceCooldownCurveSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&monitoredIndex, sizeof(int));
	hipMalloc((void**)&timeDifference, sizeof(double));
	hipMalloc((void**)&dX, sizeof(double));
	hipMalloc((void**)&tK, sizeof(double));
	hipMalloc((void**)&heatConductivitiesSizeRows, sizeof(int));
	hipMalloc((void**)&heatConductivities, _heatConductivitiesSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&specificHeatsSizeRows, sizeof(int));
	hipMalloc((void**)&specificHeats, _specificHeatsSizeRows[0] * 2 * sizeof(double));
	hipMalloc((void**)&density, sizeof(double));
	hipMalloc((void**)&htcValues, _dimensionNumber[0] * 2 * sizeof(double));
	hipMalloc((void**)&currentTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&previousTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));
	hipMalloc((void**)&g, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double));

	// Copy host variable values to device variables.
	hipMemcpy(dimensionNumber2, _dimensionNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleBestFitness2, _particleBestFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particleBestPosition2, _particleBestPosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(particlePosition2, _particlePosition, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(rangeMin2, _rangeMin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rangeMax2, _rangeMax, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(particleFitness, _particleFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(initialTemperature, _initialTemperature, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(horizontalSplitting, _horizontalSplitting, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(referenceCooldownCurveSizeRows, _referenceCooldownCurveSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(referenceCooldownCurve, _referenceCooldownCurve, _referenceCooldownCurveSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(monitoredIndex, _monitoredIndex, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(timeDifference, _timeDifference, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dX, _dX, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(tK, _tK, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(heatConductivitiesSizeRows, _heatConductivitiesSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(heatConductivities, _heatConductivities, _heatConductivitiesSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(specificHeatsSizeRows, _specificHeatsSizeRows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(specificHeats, _specificHeats, _specificHeatsSizeRows[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(density, _density, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(htcValues, _htcValues, _dimensionNumber[0] * 2 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(currentTemperature, _currentTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(previousTemperature, _previousTemperature, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(g, _g, _horizontalSplitting[0] * _particleSwarmSize[0] * sizeof(double), hipMemcpyHostToDevice);

	// Free memory.
	delete[] _currentTemperature;
	delete[] _previousTemperature;
	delete[] _g;

	// Call kernel function.
	ObjectiveFunctionKernel << <1, _particleSwarmSize[0] >> > (dimensionNumber2,
		particleBestFitness2, particleBestPosition2, particlePosition2,
		rangeMin2, rangeMax2, particleFitness, initialTemperature, horizontalSplitting,
		referenceCooldownCurveSizeRows, referenceCooldownCurve, monitoredIndex,
		timeDifference, dX, tK, heatConductivitiesSizeRows, heatConductivities,
		specificHeatsSizeRows, specificHeats, density, htcValues,
		currentTemperature, previousTemperature, g);

	_cudaStatus = hipGetLastError();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << hipGetErrorString(_cudaStatus);
		_cudaError += "ObjectiveFunctionKernel launch failed: " + ss.str() + "\n";
		fprintf(stderr, "ObjectiveFunctionKernel launch failed: %s\n", hipGetErrorString(_cudaStatus));
	}
	// Wait for all threads to finish.
	_cudaStatus = hipDeviceSynchronize();
	if (_cudaStatus != hipSuccess) {
		std::stringstream ss;
		ss << _cudaStatus;
		_cudaError += "hipDeviceSynchronize returned error code " + ss.str() + " after launching ObjectiveFunctionKernel!\n";
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching ObjectiveFunctionKernel!\n", _cudaStatus);
	}

	// Copy device variable values to host variables.
	hipMemcpy(_particleBestFitness, particleBestFitness2, _particleSwarmSize[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleBestPosition, particleBestPosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particlePosition, particlePosition2, _particleSwarmSize[0] * _dimensionNumber[0] * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(_particleFitness, particleFitness, _particleSwarmSize[0] * sizeof(double), hipMemcpyDeviceToHost);

	// Free device variables.
	hipFree(dimensionNumber2);
	hipFree(particleBestFitness2);
	hipFree(particleBestPosition2);
	hipFree(particlePosition2);
	hipFree(rangeMin2);
	hipFree(rangeMax2);
	hipFree(particleFitness);
	hipFree(initialTemperature);
	hipFree(horizontalSplitting);
	hipFree(referenceCooldownCurveSizeRows);
	hipFree(referenceCooldownCurve);
	hipFree(monitoredIndex);
	hipFree(timeDifference);
	hipFree(dX);
	hipFree(tK);
	hipFree(heatConductivitiesSizeRows);
	hipFree(heatConductivities);
	hipFree(specificHeatsSizeRows);
	hipFree(specificHeats);
	hipFree(density);
	hipFree(htcValues);
	hipFree(currentTemperature);
	hipFree(previousTemperature);
	hipFree(g);
	//hipDeviceReset();

#pragma endregion
}

void Optimize()
{
	int staticEpochs = 0;

	while (_epoch[0] < _maxEpochs[0] && staticEpochs < _maxStaticEpochs[0])
	{
		double* beta = new double[1];
		beta[0] = (0.9 - 0.55) * (_maxEpochs[0] - _epoch[0]) / _maxEpochs[0] + 0.55;
		bool isErrorImproved = false;
		double* mBest = new double[1];
		mBest[0] = GetSwarmAverageBestPosition();
		OptimizePosition(beta, mBest);

		if (_cudaStatus != hipSuccess)
		{
			return;
		}

		//WriteIterationResult(_epoch[0]);
		double improovedFitness = 0;
		double* improovedPosition = new double[_dimensionNumber[0]];

		for (int i = 0; i < _particleSwarmSize[0]; i += 1)
		{
			if (_particleFitness[i] < _globalBestFitness[_globalBestSize[0] - 1])
			{
				improovedFitness = _particleBestFitness[i];

				for (int j = 0; j < _dimensionNumber[0]; j += 1)
				{
					improovedPosition[j] = _particleBestPosition[(i * _dimensionNumber[0]) + j];
				}

				isErrorImproved = true;
				staticEpochs = 0;
			}
		}

		if (!isErrorImproved)
		{
			staticEpochs += 1;
		}
		else
		{
			double* tempFitness = new double[_globalBestSize[0]];
			double* tempPosition = new double[_globalBestSize[0] * _dimensionNumber[0]];

			for (int i = 0; i < _globalBestSize[0]; i += 1)
			{
				tempFitness[i] = _globalBestFitness[i];

				for (int j = 0; j < _dimensionNumber[0]; j += 1)
				{
					tempPosition[(i * _dimensionNumber[0]) + j] = _globalBestPosition[(i * _dimensionNumber[0]) + j];
				}
			}

			delete[] _globalBestFitness;
			delete[] _globalBestPosition;
			_globalBestSize[0] += 1;
			_globalBestFitness = new double[_globalBestSize[0]];
			_globalBestPosition = new double[_globalBestSize[0] * _dimensionNumber[0]];

			for (int i = 0; i < _globalBestSize[0]; i += 1)
			{
				if (i < _globalBestSize[0] - 1)
				{
					_globalBestFitness[i] = tempFitness[i];

					for (int j = 0; j < _dimensionNumber[0]; j += 1)
					{
						_globalBestPosition[(i * _dimensionNumber[0]) + j] = tempPosition[(i * _dimensionNumber[0]) + j];
					}
				}
				else
				{
					_globalBestFitness[i] = improovedFitness;
					std::string positions = "New best found at iteration: ";
					std::stringstream ss1;
					ss1 << _epoch[0];
					positions += ss1.str() + ", with value of: ";
					std::stringstream ss2;
					ss2 << improovedFitness;
					positions += ss2.str() + ", at ";

					for (int j = 0; j < _dimensionNumber[0]; j += 1)
					{
						std::stringstream ss3;
						ss3 << (j + 1);
						std::stringstream ss4;
						ss4 << improovedPosition[j];
						positions += ("position" + ss3.str() + ": " + ss4.str() + " ");
						_globalBestPosition[(i * _dimensionNumber[0]) + j] = improovedPosition[j];
					}

					positions += "\r\n";
					printf(positions.c_str());
				}
			}

			delete[] tempFitness;
			delete[] tempPosition;

			if (fabs(_globalBestFitness[_globalBestSize[0] - 1]) <= _particleEpsilon[0])
			{
				std::stringstream ss;
				ss << _particleEpsilon[0];
				_exitReason = "The particle swarm optimization reached an acceptable fitness value. The value was given at: " + ss.str();

				return;
			}
		}

		delete[] improovedPosition;
		_epoch[0] += 1;

		if (_globalBestSize[0] >= 5)
		{
			if ((_globalBestFitness[_globalBestSize[0] - 1] / _globalBestFitness[_globalBestSize[0] - 5]) < 0.003)
			{
				_exitReason = "The PSO global fitness value changed less then 0.003% over the last 5 global value refresh.";

				return;
			}
		}

		if (((double)_globalBestSize[0] / (double)_epoch[0]) < 0.03)
		{
			std::string value = "";

			switch (_particleOptimalisationType[0])
			{
			case 1:
				value = "Clerc";
				break;
			case 2:
				value = "Quantum";
				break;
			}

			_exitReason = "The convergence of the " + value + " PSO is too slow, it may not find the optimum";

			return;
		}

		if (staticEpochs >= _maxStaticEpochs[0])
		{
			std::stringstream ss;
			ss << _maxStaticEpochs[0];
			_exitReason = "Static iteration limit reached, limit was: " + ss.str();

			return;
		}

		if (_epoch[0] >= _maxEpochs[0])
		{
			std::stringstream ss;
			ss << _maxEpochs[0];
			_exitReason = "Iteration limit reached, limit was: " + ss.str();

			return;
		}
	}
}

void UpdateRing()
{
	int* particleIndex = Shuffle(GetIntegerRange(0, _particleSwarmSize[0]), _particleSwarmSize[0]);

	for (int i = 0; i < _particleSwarmSize[0]; i += 1)
	{
		if (_particleInformers == NULL)
		{
			_particleInformers = new int[_particleSwarmSize[0] * _particleInformerNumber[0]];
		}

		int numberOfinformers = (_particleInformerNumber[0] / 2);
		int currentInformer = 0;

		for (int n = 1; n <= numberOfinformers; n += 1)
		{
			int p = (i - n);

			if (p < 0)
			{
				p = (_particleSwarmSize[0] + p);
			}

			_particleInformers[(i * _particleInformerNumber[0]) + currentInformer] = particleIndex[p];
			currentInformer += 1;
		}

		numberOfinformers += (_particleInformerNumber[0] % 2);

		for (int n = 1; n <= numberOfinformers; n += 1)
		{
			int p = (i + n);

			if (p >= _particleSwarmSize[0])
			{
				p = (p - _particleSwarmSize[0]);
			}

			_particleInformers[(i * _particleInformerNumber[0]) + currentInformer] = particleIndex[p];
			currentInformer += 1;
		}
	}

	delete[] particleIndex;
}

int* GetIntegerRange(int startIndex, int count)
{
	int* returnValue = new int[count];

	for (int i = 0; i < count; i += 1)
	{
		returnValue[i] = (startIndex + i);
	}

	return returnValue;
}

int* Shuffle(int* index, int indexSize)
{
	for (int i = 0; i < indexSize; i += 1)
	{
		int n = ((indexSize - i) * ((double)rand() / RAND_MAX) + i);

		if (n >= 100)
		{
			n = 99;
		}

		int temp = index[n];
		index[n] = index[i];
		index[i] = temp;
	}

	return index;
}

double GetSwarmAverageBestPosition()
{
	double sum = 0;

	for (int i = 0; i < _particleSwarmSize[0]; i += 1)
	{
		for (int j = 0; j < _dimensionNumber[0]; j += 1)
		{
			sum += _particleBestPosition[(i * _dimensionNumber[0]) + j];
		}
	}

	return ((sum * 1.0) / _particleSwarmSize[0]);
}

std::vector<std::string> explode(std::string originalString, char delimeter)
{
	std::vector<std::string> elems;
	split(originalString, delimeter, std::back_inserter(elems));

	return elems;
}

template<typename Out>
void split(std::string content, char delimeter, Out result)
{
	std::stringstream ss(content);
	std::string item = "";

	while (std::getline(ss, item, delimeter))
	{
		*(result++) = item;
	}
}

class Particle
{
public:
	double* position;
	double* velocity;
	int index;
	double fitness;
	double* bestPosition;
	double bestFitness;
};

#pragma endregion